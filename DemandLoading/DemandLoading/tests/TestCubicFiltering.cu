#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <OptiXToolkit/Error/cudaErrorCheck.h>
#include <OptiXToolkit/DemandLoading/DeviceContext.h>
#include <OptiXToolkit/DemandLoading/Texture2D.h>
#include <OptiXToolkit/DemandLoading/Texture2DCubic.h>

#include "TestSparseTexture.h"

using namespace demandLoading;

__global__ static void cubicTextureDrawKernel( demandLoading::DeviceContext context, unsigned int textureId,
                                          float4* output, int width, int height, float2 ddx, float2 ddy )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    bool resident = true;
    float s = (x + 0.5f) / width;
    float t = (y + 0.5f) / height;
    float4 color;
    resident = textureUdim<float4>( context, textureId, s, t, ddx, ddy, &color );

    output[y * width + x] = resident ? color : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchCubicTextureDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context, unsigned int textureId,
                                            float4* output, int width, int height, float2 ddx, float2 ddy )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    cubicTextureDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, output, width, height, ddx, ddy );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__device__ __forceinline__ float mix( float a, float b, float x )
{
    return (1.0f-x)*a + x*b;
}

__global__ static void cubicTextureSubimageDrawKernel( demandLoading::DeviceContext context,
                                                       unsigned int textureId, float4* image, float4* drdsImage, int width, int height,
                                                       float2 uv00, float2 uv11, float2 ddx, float2 ddy )
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if( i >= width || j >= height )
        return;

    float x = ( i + 0.5f ) / width;
    float y = ( j + 0.5f ) / height;
    float s = mix( uv00.x, uv11.x, x );
    float t = mix( uv00.y, uv11.y, y );

    float4 val, drds, drdt;
    textureUdim<float4>( context, textureId, s, t, ddx, ddy, &val, &drds, &drdt );

    int pixelId = j * width + i;
    image[pixelId] = val;
    drdsImage[pixelId] = float4{drds.x, drdt.x, 0.0f, 0.0f};
}

__host__ void launchCubicTextureSubimageDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context,
                                                    unsigned int textureId, float4* image, float4* drdsImage, int width, int height,
                                                    float2 uv00, float2 uv11, float2 ddx, float2 ddy )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    cubicTextureSubimageDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, image, drdsImage, width, height, uv00, uv11, ddx, ddy );
    OTK_ERROR_CHECK( hipGetLastError() );
}
