#include "hip/hip_runtime.h"
//
// Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <OptiXToolkit/Error/cudaErrorCheck.h>
#include <OptiXToolkit/DemandLoading/DeviceContext.h>
#include <OptiXToolkit/DemandLoading/Texture2D.h>
#include <OptiXToolkit/DemandLoading/Texture2DCubic.h>

#include "TestSparseTexture.h"

using namespace demandLoading;

__global__ static void sparseTextureKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    float s = x / (float)width;
    float t = y / (float)height;

    bool   isResident = true;
#ifdef SPARSE_TEX_SUPPORT
    float4 pixel = tex2DLod<float4>( texture, s, t, lod, &isResident );
#else
    float4 pixel = tex2DLod<float4>( texture, s, t, lod );
#endif

    output[y * width + x] = isResident ? pixel : make_float4( -1.f, -1.f, -1.f, -1.f );
}

__host__ void launchSparseTextureKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    sparseTextureKernel<<<dimGrid, dimBlock>>>( texture, output, width, height, lod );
    OTK_ERROR_CHECK( hipGetLastError() );
}

__global__ static void wrapTestKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    // Test with s/t from [-1:2]
    float s = 3.f * x / (float)width - 1;
    float t = 3.f * y / (float)height - 1;

    bool isResident = true;
#ifdef SPARSE_TEX_SUPPORT
    float4 pixel = tex2DLod<float4>( texture, s, t, lod, &isResident );
#else
    float4 pixel = tex2DLod<float4>( texture, s, t, lod );
#endif

    output[y * width + x] = isResident ? pixel : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchWrapTestKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    wrapTestKernel<<<dimGrid, dimBlock>>>( texture, output, width, height, lod );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__global__ static void textureDrawKernel( demandLoading::DeviceContext context, unsigned int textureId,
                                          float4* output, int width, int height )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    bool resident = true;
    float s = (x + 0.5f) / width;
    float t = (y + 0.5f) / height;
    float2 ddx = float2{ 1.0f / width, 0.0f };
    float2 ddy = float2{ 0.0f, 1.0f / height };
    float4 color = tex2DGrad<float4>( context, textureId, s, t, ddx, ddy, &resident );

    output[y * width + x] = resident ? color : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchTextureDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context, unsigned int textureId,
                                       float4* output, int width, int height )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    textureDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, output, width, height );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__global__ static void cubicTextureDrawKernel( demandLoading::DeviceContext context, unsigned int textureId,
                                          float4* output, int width, int height, float2 ddx, float2 ddy )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    bool resident = true;
    float s = (x + 0.5f) / width;
    float t = (y + 0.5f) / height;
    float4 color;
    resident = textureUdim<float4>( context, textureId, s, t, ddx, ddy, &color );

    output[y * width + x] = resident ? color : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchCubicTextureDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context, unsigned int textureId,
                                            float4* output, int width, int height, float2 ddx, float2 ddy )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    cubicTextureDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, output, width, height, ddx, ddy );
    OTK_ERROR_CHECK( hipGetLastError() );
}
