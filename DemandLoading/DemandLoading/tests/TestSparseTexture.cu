#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <OptiXToolkit/Error/cudaErrorCheck.h>
#include <OptiXToolkit/DemandLoading/DeviceContext.h>
#include <OptiXToolkit/DemandLoading/Texture2D.h>
#include <OptiXToolkit/DemandLoading/Texture2DCubic.h>

#include "TestSparseTexture.h"

using namespace demandLoading;

__global__ static void sparseTextureKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    float s = x / (float)width;
    float t = y / (float)height;

    bool   isResident = true;
#ifdef SPARSE_TEX_SUPPORT
    float4 pixel = tex2DLod<float4>( texture, s, t, lod, &isResident );
#else
    float4 pixel = tex2DLod<float4>( texture, s, t, lod );
#endif

    output[y * width + x] = isResident ? pixel : make_float4( -1.f, -1.f, -1.f, -1.f );
}

__host__ void launchSparseTextureKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    sparseTextureKernel<<<dimGrid, dimBlock>>>( texture, output, width, height, lod );
    OTK_ERROR_CHECK( hipGetLastError() );
}

__global__ static void wrapTestKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    // Test with s/t from [-1:2]
    float s = 3.f * x / (float)width - 1;
    float t = 3.f * y / (float)height - 1;

    bool isResident = true;
#ifdef SPARSE_TEX_SUPPORT
    float4 pixel = tex2DLod<float4>( texture, s, t, lod, &isResident );
#else
    float4 pixel = tex2DLod<float4>( texture, s, t, lod );
#endif

    output[y * width + x] = isResident ? pixel : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchWrapTestKernel( hipTextureObject_t texture, float4* output, int width, int height, float lod )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    wrapTestKernel<<<dimGrid, dimBlock>>>( texture, output, width, height, lod );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__global__ static void textureDrawKernel( demandLoading::DeviceContext context, unsigned int textureId,
                                          float4* output, int width, int height )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    bool resident = true;
    float s = (x + 0.5f) / width;
    float t = (y + 0.5f) / height;
    float2 ddx = float2{ 1.0f / width, 0.0f };
    float2 ddy = float2{ 0.0f, 1.0f / height };
    float4 color = tex2DGrad<float4>( context, textureId, s, t, ddx, ddy, &resident );

    output[y * width + x] = resident ? color : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchTextureDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context, unsigned int textureId,
                                       float4* output, int width, int height )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    textureDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, output, width, height );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__global__ static void cubicTextureDrawKernel( demandLoading::DeviceContext context, unsigned int textureId,
                                          float4* output, int width, int height, float2 ddx, float2 ddy )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if( x >= width || y >= height )
        return;

    bool resident = true;
    float s = (x + 0.5f) / width;
    float t = (y + 0.5f) / height;
    float4 color;
    resident = textureUdim<float4>( context, textureId, s, t, ddx, ddy, &color );

    output[y * width + x] = resident ? color : make_float4( 1.f, 0.f, 1.f, 0.f );
}

__host__ void launchCubicTextureDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context, unsigned int textureId,
                                            float4* output, int width, int height, float2 ddx, float2 ddy )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    cubicTextureDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, output, width, height, ddx, ddy );
    OTK_ERROR_CHECK( hipGetLastError() );
}


__device__ __forceinline__ float mix( float a, float b, float x )
{
    return (1.0f-x)*a + x*b;
}

__global__ static void cubicTextureSubimageDrawKernel( demandLoading::DeviceContext context,
                                                       unsigned int textureId, float4* image, float4* drdsImage, int width, int height,
                                                       float2 uv00, float2 uv11, float2 ddx, float2 ddy )
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if( i >= width || j >= height )
        return;

    float x = ( i + 0.5f ) / width;
    float y = ( j + 0.5f ) / height;
    float s = mix( uv00.x, uv11.x, x );
    float t = mix( uv00.y, uv11.y, y );

    float4 val, drds, drdt;
    textureUdim<float4>( context, textureId, s, t, ddx, ddy, &val, &drds, &drdt );

    int pixelId = j * width + i;
    image[pixelId] = val;
    drdsImage[pixelId] = float4{drds.x, drdt.x, 0.0f, 0.0f};
}

__host__ void launchCubicTextureSubimageDrawKernel( hipStream_t stream, demandLoading::DeviceContext& context,
                                                    unsigned int textureId, float4* image, float4* drdsImage, int width, int height,
                                                    float2 uv00, float2 uv11, float2 ddx, float2 ddy )
{
    dim3 dimBlock( 16, 16 );
    dim3 dimGrid( ( width + dimBlock.x - 1 ) / dimBlock.x, ( height + dimBlock.y - 1 ) / dimBlock.y );
    cubicTextureSubimageDrawKernel<<<dimGrid, dimBlock, 0U, stream>>>( context, textureId, image, drdsImage, width, height, uv00, uv11, ddx, ddy );
    OTK_ERROR_CHECK( hipGetLastError() );
}
