#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2021-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <hipcub/hipcub.hpp>

#include <OptiXToolkit/CuOmmBaking/CuOmmBaking.h>

struct Or
{
    /// logical or operator, returns <tt>a | b</tt>
    template <typename T>
    __host__ __device__ __forceinline__ T operator()( const T& a, const T& b ) const
    {
        return a | b;
    }
};

// Bake the luminance texture to a state table, to be consumed by cuOmmBaking
__global__ void __launch_bounds__( 128 ) bakeLuminanceOpacity(
    uint32_t width,
    uint32_t height,
    uint32_t pitchInBytes,
    float transparencyCutoff,
    float opacityCutoff,
    hipTextureObject_t texture,
    uint64_t* output )
{
    const uint32_t lane = threadIdx.x;
    const uint32_t warp = threadIdx.y;

    const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

    if( y < height )
    {
        const float dx = 1 / ( float )width;
        const float dy = 1 / ( float )height;

        const float u = x * dx;
        const float v = y * dy;

        // load the luminance from the texture
        const float4 color = tex2D<float4>( texture, u, v );
        const float luminance = 0.2126f * color.x + 0.7152f * color.y + 0.0722f * color.z;

        cuOmmBaking::OpacityState state = cuOmmBaking::OpacityState::STATE_UNKNOWN;
        if( luminance <= transparencyCutoff )
            state = cuOmmBaking::OpacityState::STATE_TRANSPARENT;
        else if( luminance >= opacityCutoff )
            state = cuOmmBaking::OpacityState::STATE_OPAQUE;

        // pack a warp worth of states into 64 bit using warp reduction and have a single thread write it out.
        uint64_t mask = ( ( uint64_t )state ) << ( ( x % 32 ) * 2 );

        typedef hipcub::WarpReduce<uint64_t> WarpReduce;

        // Allocate WarpReduce shared memory for 4 warps.
        __shared__ typename WarpReduce::TempStorage temp_storage[4];

        uint64_t aggregate = WarpReduce( temp_storage[warp] ).Reduce<Or>( mask, Or() );

        // The first lane writes out the packed 64 bit warp state vector.
        if( lane == 0 )
        {
            uint32_t byte = x / 4;
            *( uint64_t* )( ( uint64_t )output + byte + y * pitchInBytes ) = aggregate;
        }
    }
}

__host__ hipError_t launchBakeLuminanceOpacity( 
    uint32_t width,
    uint32_t height,
    uint32_t pitchInBytes,
    float transparencyCutoff,
    float opacityCutoff,
    hipTextureObject_t texture,
    uint64_t* output )
{
    // the output is padded so rows are a multiple of 32 texels.
    if( ( pitchInBytes % 8 ) != 0 )
        return hipErrorInvalidPitchValue;

    if( pitchInBytes * 8 < width * 2 )
        return hipErrorInvalidValue;

    if( opacityCutoff < transparencyCutoff )
        return hipErrorInvalidValue;

    dim3 threadsPerBlock( 32, 4 );
    dim3 blocksPeGrid(
        ( uint32_t )( ( width + threadsPerBlock.x - 1 ) / threadsPerBlock.x ),
        ( uint32_t )( ( height + threadsPerBlock.y - 1 ) / threadsPerBlock.y ),
        1 );
    if( width && height )
        bakeLuminanceOpacity << <blocksPeGrid, threadsPerBlock, 0 >> > ( width, height, pitchInBytes, transparencyCutoff, opacityCutoff, texture, output );

    return hipSuccess;
}
