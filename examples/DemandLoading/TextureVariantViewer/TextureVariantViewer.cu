#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2021-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <OptiXToolkit/ShaderUtil/ray_cone.h>
#include <OptiXToolkit/ShaderUtil/vec_math.h>

#include <OptiXToolkit/DemandTextureAppBase/LaunchParams.h>
#include <OptiXToolkit/DemandTextureAppBase/DemandTextureAppDeviceUtil.h>

using namespace demandLoading;
using namespace demandTextureApp;
using namespace otk;  // for vec_math operators

//------------------------------------------------------------------------------
// Params - globally visible struct
//------------------------------------------------------------------------------

extern "C" {
__constant__ Params params;
}

//------------------------------------------------------------------------------
// Ray Payload - per ray data for closest hit program
//------------------------------------------------------------------------------

struct RayPayload
{
    float4 color; 
    RayCone rayCone;
};

//------------------------------------------------------------------------------
// Optix programs
//------------------------------------------------------------------------------
 
extern "C" __global__ void __raygen__rg()
{
    uint2 px = getPixelIndex( params.num_devices, params.device_idx );
    if( !pixelInBounds( px, params.image_dim ) )
        return;

    // Eye ray
    float3 origin, direction;
    makeEyeRayOrthographic( params.camera, params.image_dim, float2{px.x+0.5f, px.y+0.5f}, origin, direction );

    // Ray payload with ray cone for orthographic view
    RayPayload payload;
    payload.color = make_float4( 0.0f );
    payload.rayCone = initRayConeOrthoCamera( params.camera.U, params.camera.V, params.image_dim );
    
    // Trace the ray
    float tmin = 0.0f;
    float tmax = 1e16f;
    traceRay( params.traversable_handle, origin, direction, tmin, tmax, OPTIX_RAY_FLAG_NONE, &payload );

    // Blend result of ray trace with tile display color
    float4 tcolor = tileDisplayColor( params.demand_texture_context, params.display_texture_id, 10, 10, px );
    float4 color  = ( 1.0f - tcolor.w ) * payload.color + tcolor.w * tcolor;

    // Put the final color in the result buffer
    params.result_buffer[px.y * params.image_dim.x + px.x] = make_color( color );
}

extern "C" __global__ void __miss__ms()
{
    // Copy miss color to ray payload
    MissData* missData = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    getRayPayload()->color = missData->background_color;
}

extern "C" __global__ void __intersection__is()
{
    const float3 origin    = optixGetObjectRayOrigin();
    const float3 direction = optixGetObjectRayDirection();

    // Intersect ray with unit square on xy plane
    float  t = -origin.z / direction.z;
    float3 p = origin + t * direction;
    float3 n = make_float3( 0.0f, 0.0f, 1.0f );

    if( t > optixGetRayTmin() && t < optixGetRayTmax() && p.x >= 0.0f && p.x <= 1.0f && p.y >= 0.0f && p.y <= 1.0f )
        optixReportIntersection( t, 0, float3_as_uints( p ), float3_as_uints( n ) );
}

extern "C" __global__ void __closesthit__ch()
{
    // The hit group data has the demand texture id.
    HitGroupData* hitData   = reinterpret_cast<HitGroupData*>( optixGetSbtDataPointer() );
    unsigned int  textureId = hitData->texture_id;

    // The hit object is a unit square, so the texture coord is the same as the hit point.
    float2 uv = make_float2( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ) );
    
    // The world space texture derivatives for a unit square that spans (0,0) to (1,1) are
    // in texture space are just dPds=(1,0,0) and dPdt=(0,1,0). 
    float dPds_len = 1.0f;
    float dPdt_len = 1.0f;

    // Get the world space ray cone width at the intersection point
    RayPayload* payload = getRayPayload();
    float rayDistance = optixGetRayTmax();
    payload->rayCone = propagate( payload->rayCone, rayDistance );
    
    // Get the texture footprint to sample from the cone width
    float footprintWidth = texFootprintWidth( payload->rayCone.width, dPds_len, dPdt_len );
    float2 ddx = make_float2( footprintWidth, 0.0f );
    float2 ddy = make_float2( 0.0f, footprintWidth );

    // Use the variant texture on the right half of the image
    uint2 px = getPixelIndex( params.num_devices, params.device_idx );
    if( px.x * 2 > params.image_dim.x )
        textureId += 1;

    // Sample the texture or variant, and return this value in the ray payload
    bool resident  = false;
    payload->color = tex2DGrad<float4>( params.demand_texture_context, textureId, uv.x, uv.y, ddx, ddy, &resident );
}
