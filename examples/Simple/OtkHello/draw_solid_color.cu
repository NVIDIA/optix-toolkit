#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//

#include "hello.h"

#include <OptiXToolkit/ShaderUtil/color.h>

#include <optix.h>

extern "C" {
__constant__ Params params;
}

extern "C"
__global__ void __raygen__draw_solid_color()
{
    uint3 launch_index = optixGetLaunchIndex();
    RayGenData* rtData = (RayGenData*)optixGetSbtDataPointer();
    params.image[launch_index.y * params.image_width + launch_index.x] =
        make_color( make_float3( rtData->r, rtData->g, rtData->b ) );
}
